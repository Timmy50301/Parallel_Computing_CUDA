#include "hip/hip_runtime.h"
#include "parameters.h"
#include <hiprand/hiprand_kernel.h> //cuda random generator

__global__ void setup(hiprandState *states, int seed_)
{
	int id=(blockIdx.x*blockDim.x+threadIdx.x);
	int seed = id*seed_;
	hiprand_init(seed,id,0,&states[id]);
};

__global__ void cuda_kernel(float *A, int ppt, hiprandState *states)
{	
	// int thread_num=blockDim.x*gridDim.x;
	int id=(blockIdx.x*blockDim.x+threadIdx.x);
	// int seed = id;
	// hiprand_init(seed,id,0,&states[id]);
	for(int i=0;i<ppt;i++){
		float x=hiprand_uniform(&states[id]); // range excludes 0.0 but includes 1.0
		float y=hiprand_uniform(&states[id]);
		// float x=0.5;
		// float y=0.5;
		float temp = (x-0.5)*(x-0.5)+(y-0.5)*(y-0.5);
		if(temp<=0.25) A[id]++;
	}
};

float GPU_kernel(float *A, int ppt, int grid_num, int thread_num, int seed){
	const int SIZE = grid_num*thread_num;
	float *dA;
	hiprandState *dev_random;

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Allocate Memory Space on Device
	hipMalloc((void**)&dA,sizeof(float)*SIZE);
	hipMalloc((void**)&dev_random,SIZE*sizeof(hiprandState));

	// Copy Data to be Calculated
	hipMemcpy(dA, A, sizeof(float)*SIZE, hipMemcpyHostToDevice);

	// Lunch SETUP Kernel
	dim3 dimGrid(grid_num);
	dim3 dimBlock(thread_num);
	setup<<<dimGrid,dimBlock>>>(dev_random,seed);

	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	cuda_kernel<<<dimGrid,dimBlock>>>(dA,ppt,dev_random);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop);

	// Copy Output back
	hipMemcpy(A, dA, sizeof(float)*SIZE, hipMemcpyDeviceToHost);

	// Release Memory Space on Device
	hipFree(dA);

	// Calculate Elapsed Time
  	float usetime; 
  	hipEventElapsedTime(&usetime, start, stop);  

	return usetime;
}

